#include "Timer.cuh"

Timer::Timer(): startEvent(), stopEvent() {
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
}

Timer::~Timer() {
    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);
}

void Timer::start() {
    hipEventRecord(startEvent);
    hipEventSynchronize(startEvent);
}

float Timer::stop() {
    hipEventRecord(stopEvent);
    hipEventSynchronize(stopEvent);
    float timeElapsed = 0.0f;
    hipEventElapsedTime(&timeElapsed, startEvent, stopEvent);
    return timeElapsed;
}
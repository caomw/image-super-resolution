#include <cmath>
#include <fstream>

#include "Matrix.cuh"

HostMatrix::HostMatrix(unsigned int height, unsigned int width): Matrix(height, width) {
    hipHostMalloc(&elements, height * width * sizeof(float));
}

HostMatrix::~HostMatrix() {
    hipHostFree(elements);
}

float HostMatrix::getElement(unsigned int i, unsigned int j) const {
    if (i < height && j < width) {
        return elements[i * width + j];
    } else {
        return NAN;
    }
}

DeviceMatrix::DeviceMatrix(unsigned int height, unsigned int width): Matrix(height, width) {
    hipMalloc(&elements, height * width * sizeof(float));
}

DeviceMatrix::~DeviceMatrix() {
    hipFree(elements);
}

float DeviceMatrix::getElement(unsigned int i, unsigned int j) const {
    if (i < height && j < width) {
        float elem = 0.0f;
        hipMemcpy(&elem, &elements[i * width + j], sizeof(float), hipMemcpyDeviceToHost);
        return elem;
    } else {
        return NAN;
    }
}

HostMatrix MatrixUtilities::loadFromFile(const char* fileName) {
    std::ifstream fin;
    fin.open(fileName);

    unsigned int height = 0;
    unsigned int width = 0;
    fin >> height >> width;

    HostMatrix matrix(height, width);
    for (unsigned int i = 0; i < height * width; ++i) {
        fin >> matrix.elements[i];
    }

    fin.close();
    return matrix;
}

HostMatrix MatrixUtilities::copyToHost(const DeviceMatrix& matrix) {
    HostMatrix copy_matrix(matrix.height, matrix.width);
    hipMemcpy(copy_matrix.elements, matrix.elements, matrix.height * matrix.width * sizeof(float), hipMemcpyDeviceToHost);
    return copy_matrix;
}

DeviceMatrix MatrixUtilities::copyToDevice(const HostMatrix& matrix) {
    DeviceMatrix copy_matrix(matrix.height, matrix.width);
    hipMemcpy(copy_matrix.elements, matrix.elements, matrix.height * matrix.width * sizeof(float), hipMemcpyHostToDevice);
    return copy_matrix;
}